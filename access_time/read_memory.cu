#include <hip/hip_runtime.h>
#include <iostream>


__global__ void readMemory(int *d_array, int size, int *output) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        int start = clock(); 
        volatile  int temp = d_array[idx]; 
        int end = clock();
        output[idx] = end - start + temp; 
    }
}

int main() {
    int size = 1; 
    int *h_array = new int[size];
    int *h_output = new int[size];

    
    for (int i = 0; i < size; i++) {
        h_array[i] = i;
    }

    int *d_array;
    int *d_output;

  
    hipMalloc((void**)&d_array, size * sizeof(int));
    hipMalloc((void**)&d_output, size * sizeof(int));

    
    hipMemcpy(d_array, h_array, size * sizeof(int), hipMemcpyHostToDevice);

    
    readMemory<<<1, 1>>>(d_array, size, d_output);

    
    hipMemcpy(h_output, d_output, size * sizeof(int), hipMemcpyDeviceToHost);

   
    std::cout << "Time to read memory: " << h_output[0] - h_array[0] << " clock cycles" << std::endl;

    // 释放内存
    hipFree(d_array);
    hipFree(d_output);
    delete[] h_array;
    delete[] h_output;

    return 0;
}

